#include "hip/hip_runtime.h"
#include "../include/CudaKernels.h"

// CUDA includes
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/device_functions.h>


unsigned int iDivUp(unsigned int a, unsigned int b)
{
    uint c = a/b;
    c += (a%b == 0) ? 0: 1;
    return c;
}


__global__ void cuVectorInc(float * vec, const unsigned int n)
{
    unsigned int  idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < n)
    {
        atomicAdd(&vec[idx], 1);
    }
}




void CudaKernels::VectorInc(float * vec, const unsigned int n)
{
    unsigned int blockSize = 1024;
    unsigned int gridSize = iDivUp(n, blockSize);
    cuVectorInc<<<gridSize, blockSize>>>(vec, n);
}
