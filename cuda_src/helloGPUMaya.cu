#include "hip/hip_runtime.h"
#include "../include/CudaKernels.h"

// CUDA includes
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>


unsigned int iDivUp(unsigned int a, unsigned int b)
{
    uint c = a/b;
    c += (a%b == 0) ? 0: 1;
    return c;
}


__global__ void cuVectorInc(float * vec, const unsigned int n)
{
    unsigned int  idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < n)
    {
        atomicAdd(&vec[idx], 1);
    }
}




void CudaKernels::VectorInc(float * vec, const unsigned int n)
{
    thrust::device_vector<float> d_vec(vec, vec+n);
    float * d_vec_ptr = thrust::raw_pointer_cast(&vec[0]);

    unsigned int blockSize = 1024;
    unsigned int gridSize = iDivUp(n, blockSize);
    cuVectorInc<<<gridSize, blockSize>>>(d_vec_ptr, n);
    hipDeviceSynchronize();

    thrust::copy(d_vec.begin(), d_vec.end(), vec);
}
