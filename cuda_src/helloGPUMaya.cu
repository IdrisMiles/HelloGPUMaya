#include "hip/hip_runtime.h"
#include "../include/helloGPUMaya.h"

// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

__global__ void cuVectorInc(float * vec, const uint n)
{
    uint idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < n)
    {
        atomicAdd(&vec[idx], 1);
    }
}
